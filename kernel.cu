#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "wb.h"

__global__ void vecAdd(float* in1, float* in2, float* out, int len)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < len) out[i] = in1[i] + in2[i];
}

int main(int argc, char** argv)
{
    wbArg_t args;
    int inputLength;
    float* hostInput1;
    float* hostInput2;
    float* hostOutput;
    float* deviceInput1;
    float* deviceInput2;
    float* deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 =
        (float*)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 =
        (float*)wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float*)malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

    wbTime_start(GPU, "Allocating GPU memory.");
    hipMalloc((void**)&deviceInput1, inputLength);
    hipMalloc((void**)&deviceInput2, inputLength);
    hipMalloc((void**)&deviceOutput, inputLength);

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Скопируйте память на GPU
    hipMemcpy(deviceInput1, hostInput1, inputLength, hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, inputLength, hipMemcpyHostToDevice);
    hipMemcpy(deviceOutput, hostOutput, inputLength, hipMemcpyDeviceToHost);

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    //@@ Инициализация сетки и размеры блока здесь
    dim3 DimGrid((inputLength - 1) / 256 + 1, 1, 1);
    dim3 DimBlock(256, 1, 1);
    vecAdd << <DimGrid, DimBlock >> > (deviceInput1, deviceInput2, deviceOutput, inputLength);

    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Запустите ядро GPU
    vecAdd << <ceil(inputLength / 256.0), 256 >> > (deviceInput1, deviceInput2, deviceOutput, inputLength);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Скопируйте  GPU память обратно на CPU здесь
    hipMemcpy(deviceInput1, hostInput1, inputLength, hipMemcpyDeviceToHost);
    hipMemcpy(deviceInput2, hostInput2, inputLength, hipMemcpyDeviceToHost);
    hipMemcpy(deviceOutput, hostOutput, inputLength, hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Освободите память GPU
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}